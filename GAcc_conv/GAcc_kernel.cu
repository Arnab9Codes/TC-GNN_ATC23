#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_runtime.h>

#include "config.h"

using namespace nvcuda;

#define MAX_DIM 100
#define MAX_NB 100       // must <= partsize 
#define threadPerWarp 2 //must < 32
#define wrapPerBlock 1  // must also set with respect to the 
                        // [thread-per-block = wrapPerBlock *  threadPerWarp]

__device__ inline float atomicAdd_F(float* address, float value)
{
  float old = value;  
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
}

//////////////////////
/// SPMM forward
//////////////////////
__global__ void spmm_forward_cuda_kernel(
	const int * __restrict__ nodePointer,		// node pointer.
	const int *__restrict__ edgeList,			// edge list.
	const int *__restrict__ blockPartition, 	// number of TC_blocks (16x8) in each row_window.
	const int *__restrict__ edgeToColumn, 		// eid -> col within each row_window.
	const int *__restrict__ edgeToRow, 		    // eid -> col within each row_window.
	const int numNodes,
	const int numEdges,
	const int embedding_dim,				    // embedding dimension.
	const float *__restrict__ in_mat,		    // input feature matrix.
	float *out_mat							    // aggregated output feature matrix.
);

//////////////////////
/// SDDMM forward
//////////////////////
__global__ void sddmm_forward_cuda_kernel(
    const int * __restrict__ nodePointer,		// node pointer.
	const int *__restrict__ edgeList,			// edge list.
	const int *__restrict__ blockPartition,		// number of TC_blocks (16x8) in each row_window.
	const int *__restrict__ edgeToColumn, 		// eid -> col within each row_window.
	const int *__restrict__ edgeToRow, 			// eid -> col within each row_window.
	const int numNodes,
	const int numEdges,
	const int embedding_dim,				    // embedding dimension.
	float *__restrict__ in_mat,					// input feature matrix.
	float *edgeFeature							// aggregated output feature matrix.
);


template <typename scalar_t>
__global__ void spmm_backward_cuda_kernel(
    int num_nodes, 
    int dim,
    int num_parts,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part2Node
);


////////////////////////////////////////////
//
// SPMM Foward Pass
//
////////////////////////////////////////////
std::vector<torch::Tensor> spmm_forward_cuda(
    torch::Tensor nodePointer,
    torch::Tensor edgeList,
    torch::Tensor blockPartition, 
    torch::Tensor edgeToColumn,
    torch::Tensor edgeToRow,
              int num_nodes,
              int num_edges,
              int embedding_dim,
    torch::Tensor input
) 
{
    auto output = torch::zeros_like(input);
    const int num_row_windows = blockPartition.size(0);
    const int WARPperBlock = 16;

    dim3 grid(num_row_windows, 1, 1);
    dim3 block(WARP_SIZE, WARPperBlock, 1);

    const int dimTileNum = embedding_dim / BLK_H;
	const int dynamic_shared_size = dimTileNum * BLK_W * BLK_H * sizeof(float); // dynamic shared memory.

    spmm_forward_cuda_kernel<<<grid, block, dynamic_shared_size>>>(
                                                                    nodePointer.data<int>(), 
                                                                    edgeList.data<int>(),
                                                                    blockPartition.data<int>(), 
                                                                    edgeToColumn.data<int>(), 
                                                                    edgeToRow.data<int>(), 
                                                                    num_nodes,
                                                                    num_edges,
                                                                    embedding_dim,
                                                                    input.data<float>(), 
                                                                    output.data<float>()
                                                                );

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    return {output};
}

////////////////////////////////////////////
//
// SPMM Foward Pass
//
////////////////////////////////////////////
std::vector<torch::Tensor> sddmm_forward_cuda(
    torch::Tensor nodePointer,
    torch::Tensor edgeList,			    // edge list.
	torch::Tensor blockPartition,		// number of TC_blocks (16x8) in each row_window.
	torch::Tensor edgeToColumn, 		// eid -> col within each row_window.
	torch::Tensor edgeToRow, 			// eid -> col within each row_window.
              int num_nodes,
              int num_edges,
              int embedding_dim,	    // embedding dimension.
	torch::Tensor input				    // input feature matrix.
) 
{
    auto output = torch::zeros_like(edgeList);
    const int num_row_windows = blockPartition.size(0);

	dim3 grid(num_row_windows, 1, 1);
	dim3 block(WARP_SIZE, 1, 1);

    sddmm_forward_cuda_kernel<<< grid, block>>>(
                                                nodePointer.data<int>(), 
                                                edgeList.data<int>(),
                                                blockPartition.data<int>(), 
                                                edgeToColumn.data<int>(), 
                                                edgeToRow.data<int>(), 
                                                num_nodes,
                                                num_edges,
                                                embedding_dim,
                                                input.data<float>(), 
                                                output.data<float>()
                                                );

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    return {output};
}


//////////////////////
/// SPMM forward
//////////////////////
__global__ void spmm_forward_cuda_kernel(
	const int * __restrict__ nodePointer,		// node pointer.
	const int *__restrict__ edgeList,			// edge list.
	const int *__restrict__ blockPartition, 	// number of TC_blocks (16x8) in each row_window.
	const int *__restrict__ edgeToColumn, 		// eid -> col within each row_window.
	const int *__restrict__ edgeToRow, 		    // eid -> col within each row_window.
	const int numNodes,
	const int numEdges,
	const int embedding_dim,				    // embedding dimension.
	const float *__restrict__ input,		    // input feature matrix.
	float *output							    // aggregated output feature matrix.
) {
    const unsigned bid = blockIdx.x;								// block_index == row_window_index
	const unsigned wid = threadIdx.y;								// warp_index handling multi-dimension > 16.
	const unsigned laneid = threadIdx.x;							// lanid of each warp.
	const unsigned tid = threadIdx.y * blockDim.x + laneid;			// threadid of each block.
	const unsigned warpSize = blockDim.x;							// number of threads per warp.
	const unsigned threadPerBlock = blockDim.x * blockDim.y;		// number of threads per block.

	const unsigned dimTileNum = embedding_dim / BLK_H;              // number of tiles along the dimension
	const unsigned nIdx_start = bid * BLK_H;					    // starting nodeIdx of current row_window.
	const unsigned nIdx_end = min((bid + 1) * BLK_H, numNodes);		// ending nodeIdx of current row_window.
	
	const unsigned eIdx_start = nodePointer[nIdx_start];			// starting edgeIdx of current row_window.
	const unsigned eIdx_end = nodePointer[nIdx_end];				// ending edgeIdx of the current row_window.
	const unsigned num_TC_blocks = blockPartition[bid]; 			// number of TC_blocks of the current row_window.
	const unsigned dense_bound = numNodes * embedding_dim;

	__shared__ float sparse_A[BLK_H * BLK_W];					// row-major sparse matrix shared memory store.
	__shared__ int sparse_AToX_index[BLK_W];					// TC_block col to dense_tile row.
	// __shared__ float dense_X[dimTileNum * BLK_W * BLK_H];	// column-major dense tile [dimTileNum, BLK_W, BLK_H]
	extern __shared__ float dense_X[];

	wmma::fragment<wmma::matrix_a, BLK_H, BLK_H, BLK_W, wmma::precision::tf32, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, BLK_H, BLK_H, BLK_W, wmma::precision::tf32, wmma::col_major> b_frag;
	wmma::fragment<wmma::accumulator, BLK_H, BLK_H, BLK_W, float> acc_frag;
	wmma::fill_fragment(acc_frag, 0.0f);

	// Processing TC_blocks along the column dimension of Sparse A.
	for (unsigned i = 0; i < num_TC_blocks; i++){

		// Init A_colToX_row with dummy values.
		if (tid < BLK_W){
			sparse_AToX_index[tid] = numNodes + 1;
		}

		__syncthreads();

		// Init sparse_A with zero values.
		#pragma unroll
		for (unsigned idx = tid; idx < BLK_W * BLK_H; idx += threadPerBlock){
			sparse_A[idx] = 0;
		}

		// Init dense_X with zero values.
		#pragma unroll
		for (unsigned idx = tid; idx < dimTileNum * BLK_W * BLK_H; idx += threadPerBlock){
			dense_X[idx] = 0;
		}

		// Initialize sparse_A by using BLK_H (16) threads from the warp-0.
		// currently fetch all neighbors of the current nodes.
		// then to see whether it can fit into current TC_block frame of column.		
		#pragma unroll
		for (unsigned eIdx = eIdx_start + tid; eIdx < eIdx_end; eIdx += threadPerBlock){
			unsigned col = edgeToColumn[eIdx];
			if (i * BLK_W <= col && col < (i + 1) * BLK_W){			// if the edge in the current TC_block frame of column.
				unsigned row_local = edgeToRow[eIdx] % BLK_H;
				unsigned col_local = col % BLK_W;
				sparse_A[row_local * BLK_W + col_local] = 1;		// set the edge of the sparse_A.
				sparse_AToX_index[col_local] = edgeList[eIdx];		// record the mapping from sparse_A colId to rowId of dense_X.
			}		
		}

		__syncthreads();

		// Initialize dense_X by column-major store,
		// Threads of a warp for fetching a dense_X.
		// each warp identify by wid.
		if (wid < dimTileNum)
			#pragma unroll
			for (unsigned idx = laneid; idx < BLK_W * BLK_H; idx += warpSize){
				unsigned dense_rowIdx = sparse_AToX_index[idx % BLK_W];						// TC_block_col to dense_tile_row.
				unsigned dense_dimIdx = idx / BLK_W;										// dimIndex of the dense tile.
				unsigned source_idx = dense_rowIdx * embedding_dim + wid * BLK_H + dense_dimIdx;
				unsigned target_idx = wid * BLK_W * BLK_H + idx;
				// boundary test.
				if (source_idx >= dense_bound)
					dense_X[target_idx] = 0;
				else
					dense_X[target_idx] = input[source_idx];
			}

		__syncthreads();

		if (wid < dimTileNum)
		{
			wmma::load_matrix_sync(a_frag, sparse_A, BLK_W);
			wmma::load_matrix_sync(b_frag, dense_X + wid * BLK_W * BLK_H, BLK_W);

			#pragma unroll
			for (unsigned t = 0; t < a_frag.num_elements; t++) {
				a_frag.x[t] =  wmma::__float_to_tf32(a_frag.x[t]);
			}

			#pragma unroll
			for (unsigned t = 0; t < b_frag.num_elements; t++) {
				b_frag.x[t] =  wmma::__float_to_tf32(b_frag.x[t]);
			}
			// Perform the matrix multiplication.
			wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
		}
	}

	if (wid < dimTileNum)
		// Store the matrix to output matrix.
		// * Note * embeeding dimension should be padded divisible by BLK_H for output correctness.
		wmma::store_matrix_sync(output + bid * BLK_H * embedding_dim + wid * BLK_H, acc_frag, embedding_dim, wmma::mem_row_major);
}


//////////////////////
/// SDDMM forward
//////////////////////
__global__ void sddmm_forward_cuda_kernel(
    const int *__restrict__ nodePointer,
	const int *__restrict__ edgeList,			// edge list.
	const int *__restrict__ blockPartition,		// number of TC_blocks (16x8) in each row_window.
	const int *__restrict__ edgeToColumn, 		// eid -> col within each row_window.
	const int *__restrict__ edgeToRow, 			// eid -> col within each row_window.
	const int numNodes,
	const int numEdges,
	const int embedding_dim,				    // embedding dimension.
	float *__restrict__ in_mat,					// input feature matrix.
	float *edgeFeature							// aggregated output feature matrix.
)
{
    unsigned bid = blockIdx.x;										// block_index == row_window_index
    unsigned wid = threadIdx.y;										// warp_index handling multi-dimension > 16.
    unsigned laneid = threadIdx.x;									// lanid of each warp.
    unsigned tid = threadIdx.y * blockDim.x + laneid;				// threadid of each block.

    unsigned threadPerBlock = blockDim.x * blockDim.y;
    unsigned DimIterations =  (embedding_dim + BLK_W - 1) / BLK_W; 	// dimension iteration for output.

    unsigned nid_start = bid * BLK_H;								// starting node_id of current row_window.
    unsigned nid_end = min((bid + 1) * BLK_H, numNodes);			// ending node_id of the current row_window.

    unsigned eIdx_start = nodePointer[nid_start];					            // starting eIdx of current row_window.
    unsigned eIdx_end = nodePointer[nid_end];						            // ending eIdx of the current row_window.
    unsigned num_TC_blocks = (blockPartition[bid] * BLK_W + BLK_H - 1)/BLK_H; 	// number of TC_blocks of the current row_window.

    __shared__ float sparse_A[BLK_H * BLK_H];					// 16 x 16 output sparse matrix.
    __shared__ float sparse_A_val[BLK_H * BLK_H];				// 16 x 16 output sparse matrix.
    #ifdef verify
    __shared__ float verify_A[BLK_H * BLK_H];					// 16 x 16 output sparse matrix.
    #endif 

    __shared__ unsigned sparse_AToX_index[BLK_H];				// TC_block col to dense_tile row.
    __shared__ float dense_X[BLK_H * BLK_W];
    __shared__ float dense_Y[BLK_W * BLK_H];

    wmma::fragment<wmma::matrix_a, BLK_H, BLK_H, BLK_W, wmma::precision::tf32, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, BLK_H, BLK_H, BLK_W, wmma::precision::tf32, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, BLK_H, BLK_H, BLK_W, float> acc_frag;
    wmma::fill_fragment(acc_frag, 0.0f);

    // Processing TC_blocks along the column dimension of Sparse A.
    // The block step here is 2, which is 16 = 8 + 8. 
    // In order to reuse the edgeToColumn in SpMM. 
    for (unsigned i = 0; i < num_TC_blocks; i++ ){

        if (wid == 0 && laneid < BLK_H){
            sparse_AToX_index[laneid] = numNodes + 1;
        }

        __syncthreads();

        #pragma unroll
        for (unsigned idx = tid; idx < BLK_H * BLK_H; idx += threadPerBlock){
            sparse_A[idx] = numEdges + 1;
            sparse_A_val[idx] = 0.0f;
        }

        #pragma unroll
        for (unsigned idx = tid; idx < BLK_W * BLK_H; idx += threadPerBlock){
            dense_X[idx] = 0;
            dense_Y[idx] = 0;
        }

        // Initialize sparse_A by using BLK_H (16) threads from the warp-0.
        // currently fetch all neighbors of the current nodes.
        // then to see whether it can fit into current TC_block frame of column.
        #pragma unroll
        // if (tid < WARP_SI)
        for (unsigned eIdx = tid + eIdx_start; eIdx < eIdx_end; eIdx += threadPerBlock){
            unsigned col = edgeToColumn[eIdx];						// condensed column id in sparse_A.
            if (i * BLK_H <= col && col < (i + 1) * BLK_H){			// if the edge in the current TC_block frame of column.
                unsigned row = edgeToRow[eIdx] % BLK_H;				// reverse indexing the row Id of the edge.
                sparse_A[row * BLK_H + col % BLK_H] = eIdx;			// set the edge of the sparse_A.
                sparse_AToX_index[col % BLK_H] = edgeList[eIdx];	// record the mapping from sparse_A colId to rowId of dense_X.
            }
        }		

        __syncthreads();

        for (unsigned warp_iter = 0; warp_iter < DimIterations; warp_iter++){
            // Initialize dense_X by row-major store,
            // Threads of a warp for fetching a dense_X.
            #pragma unroll
            for (unsigned i = tid; i < BLK_H * BLK_W; i += threadPerBlock){
                unsigned dense_rowIdx = i / BLK_W;					
                unsigned dense_dimIdx = i % BLK_W;					
                unsigned target_idx = i;
                unsigned source_idx = (nid_start + dense_rowIdx) * embedding_dim + warp_iter * BLK_W + dense_dimIdx;
                if (source_idx >= numNodes * embedding_dim)
                    dense_X[target_idx] = 0;
                else
                    dense_X[target_idx] = in_mat[source_idx];
            }

            // Initialize dense_Y by column-major store,
            // Threads of a warp for fetching a dense_Y.
            #pragma unroll
            for (unsigned i = tid; i < BLK_W * BLK_H; i += threadPerBlock){
                unsigned dense_rowIdx = sparse_AToX_index[i / BLK_W];					// TC_block_col to dense_tile_row.
                unsigned dense_dimIdx = i % BLK_W;										// dimIndex of the dense tile.
                unsigned target_idx = i;
                unsigned source_idx = dense_rowIdx * embedding_dim + warp_iter * BLK_W + dense_dimIdx;
                if (source_idx >= numNodes * embedding_dim)
                    dense_Y[target_idx] = 0;
                else
                    dense_Y[target_idx] = in_mat[source_idx];
            }

            __syncthreads();

            wmma::load_matrix_sync(a_frag, dense_X, BLK_W);
            wmma::load_matrix_sync(b_frag, dense_Y, BLK_W);

            #pragma unroll
            for (unsigned t = 0; t < a_frag.num_elements; t++) {
                a_frag.x[t] =  wmma::__float_to_tf32(a_frag.x[t]);
            }

            #pragma unroll
            for (unsigned t = 0; t < b_frag.num_elements; t++) {
                b_frag.x[t] =  wmma::__float_to_tf32(b_frag.x[t]);
            }

            // Perform the matrix multiplication on Tensor Core
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);		
        } // <--- ending of warp iteration.

        wmma::store_matrix_sync(sparse_A_val, acc_frag, BLK_H, wmma::mem_row_major);
        wmma::fill_fragment(acc_frag, 0.0f);

        // Output the results to sparse matrix edge featureList.
        for (unsigned t = 0; t < BLK_H * BLK_H; t++) {
            unsigned rowId = t / BLK_H;
            unsigned colId = t % BLK_H;
            if (sparse_A[rowId * BLK_H + colId] < numEdges){
                unsigned eIdx = sparse_A[rowId * BLK_H + colId];
                edgeFeature[eIdx] = sparse_A_val[t];
            }
        } //<-- ending of storing output to global memory.
    }
}

////////////////////////////////////////////
// 
// backward pass
//
////////////////////////////////////////////
std::vector<torch::Tensor> spmm_backward_cuda(
    int threadPerBlock,
    torch::Tensor d_output,
    torch::Tensor row_pointers,
    torch::Tensor column_index,
    torch::Tensor degrees,
    torch::Tensor part_pointers,
    torch::Tensor part2Node
) {

    auto d_input = torch::zeros_like(d_output);
    // d_input = d_output;
    const int dim = d_input.size(1);
    const int num_nodes = d_input.size(0);
    const int num_parts = part2Node.size(0);
    const int blocks = (num_parts * 32 + threadPerBlock - 1) / threadPerBlock; 

    AT_DISPATCH_FLOATING_TYPES(d_output.type(), "spmm_cuda_backward", ([&] {
                                spmm_backward_cuda_kernel<scalar_t><<<blocks, threadPerBlock>>>(
                                    num_nodes, 
                                    dim,
                                    num_parts,
                                    d_output.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                                    d_input.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                                    row_pointers.packed_accessor32<int,1,torch::RestrictPtrTraits>(),
                                    column_index.packed_accessor32<int,1,torch::RestrictPtrTraits>(),
                                    degrees.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
                                    part_pointers.packed_accessor32<int,1,torch::RestrictPtrTraits>(), 
                                    part2Node.packed_accessor32<int,1,torch::RestrictPtrTraits>()
                                );
                            }));
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    return {d_input};
}

template <typename scalar_t>
__global__ void spmm_backward_cuda_kernel(
    int num_nodes, 
    int dim,
    int num_parts, 
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part2Node
) {

    int tid =  blockIdx.x * blockDim.x + threadIdx.x;
    int warpId =  tid / 32;
    int intraWarp_tid = tid % 32;
    int block_warpID = threadIdx.x/32;
    
    if (warpId < num_parts && intraWarp_tid < threadPerWarp){

        __shared__  int partial_index[MAX_NB * wrapPerBlock];
        __shared__ float partial_results[MAX_DIM * wrapPerBlock];

        int srcId = part2Node[warpId];
        int partBeg = part_pointers[warpId];
        int partEnd = part_pointers[warpId + 1];
        float src_norm = degrees[srcId];

        int pindex_base = block_warpID * MAX_NB;
        for (int nid = partBeg + intraWarp_tid; nid < partEnd; nid += threadPerWarp){
            partial_index[pindex_base + nid - partBeg] = column_index[nid];
        }
         __syncthreads();

        int presult_base = block_warpID * MAX_DIM;
        for (int nid = 0; nid < partEnd - partBeg; nid++)
        {
            int nIndex = partial_index[pindex_base + nid];
            float degree_norm =  __fmaf_rn(src_norm, degrees[nIndex], 0);

            if (nid == 0)
                for (int d = intraWarp_tid; d < dim; d += threadPerWarp){
                    partial_results[presult_base + d] = 0;
                    // atomicAdd_F((float*)&d_input[srcId][d], degree_norm * d_output[nIndex][d]);
                }
            for (int d = intraWarp_tid; d < dim; d += threadPerWarp){
                partial_results[presult_base + d] += __fmaf_rn(degree_norm, d_output[nIndex][d], 0);
            }
        }
        for (int d = intraWarp_tid; d < dim; d += threadPerWarp){
            atomicAdd_F((float*)&d_input[srcId][d], partial_results[presult_base + d]);
        }
    }
}
