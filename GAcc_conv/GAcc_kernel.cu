#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_runtime.h>

#include "config.h"
#define WPB 8

using namespace nvcuda;

//////////////////////
/// SPMM forward (GCN, GraphSAGE)
//////////////////////
__global__ void spmm_forward_cuda_kernel(
	const int * __restrict__ nodePointer,		// node pointer.
	const int *__restrict__ edgeList,			// edge list.
	const int *__restrict__ blockPartition, 	// number of TC_blocks (16x8) in each row_window.
	const int *__restrict__ edgeToColumn, 		// eid -> col within each row_window.
	const int *__restrict__ edgeToRow, 		    // eid -> col within each row_window.
	const int numNodes,
	const int numEdges,
	const int embedding_dim,				    // embedding dimension.
	const float *__restrict__ in_mat,		    // input feature matrix.
	float *out_mat							    // aggregated output feature matrix.
);

//////////////////////
/// SPMM forward (GAT, AGNN)
//////////////////////
__global__ void spmmGAT_forward_cuda_kernel(
	const int * __restrict__ nodePointer,		// node pointer.
	const int *__restrict__ edgeList,			// edge list.
    const float *__restrict__ edgeAttention,	// edge attention.
	const int *__restrict__ blockPartition, 	// number of TC_blocks (16x8) in each row_window.
	const int *__restrict__ edgeToColumn, 		// eid -> col within each row_window.
	const int *__restrict__ edgeToRow, 		    // eid -> col within each row_window.
	const int numNodes,
	const int numEdges,
	const int embedding_dim,				    // embedding dimension.
	const float *__restrict__ input,		    // input feature matrix.
	float *output							    // aggregated output feature matrix.
);

//////////////////////
/// SDDMM forward
//////////////////////
__global__ void sddmm_forward_cuda_kernel(
    const int * __restrict__ nodePointer,		// node pointer.
	const int *__restrict__ edgeList,			// edge list.
	const int *__restrict__ blockPartition,		// number of TC_blocks (16x8) in each row_window.
	const int *__restrict__ edgeToColumn, 		// eid -> col within each row_window.
	const int *__restrict__ edgeToRow, 			// eid -> col within each row_window.
	const int numNodes,
	const int numEdges,
	const int embedding_dim,				    // embedding dimension.
	float *__restrict__ in_mat,					// input feature matrix.
	float *edgeFeature							// aggregated output feature matrix.
);

////////////////////////////////////////////
//
// SPMM Foward Pass  (GCN, GraphSAGE)
//
////////////////////////////////////////////
std::vector<torch::Tensor> spmm_forward_cuda(
    torch::Tensor nodePointer,
    torch::Tensor edgeList,
    torch::Tensor blockPartition, 
    torch::Tensor edgeToColumn,
    torch::Tensor edgeToRow,
              int num_nodes,
              int num_edges,
              int embedding_dim,
    torch::Tensor input
) 
{
    auto output = torch::zeros_like(input);
    const int num_row_windows = blockPartition.size(0);
    const int WARPperBlock = WPB;

    dim3 grid(num_row_windows, 1, 1);
    dim3 block(WARP_SIZE, WARPperBlock, 1);

    const int dimTileNum = (embedding_dim + BLK_H - 1) / BLK_H;
	const int dynamic_shared_size = dimTileNum * BLK_W * BLK_H * sizeof(float); // dynamic shared memory.

    spmm_forward_cuda_kernel<<<grid, block, dynamic_shared_size>>>(
                                                                    nodePointer.data<int>(), 
                                                                    edgeList.data<int>(),
                                                                    blockPartition.data<int>(), 
                                                                    edgeToColumn.data<int>(), 
                                                                    edgeToRow.data<int>(), 
                                                                    num_nodes,
                                                                    num_edges,
                                                                    embedding_dim,
                                                                    input.data<float>(), 
                                                                    output.data<float>()
                                                                );

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    return {output};
}

////////////////////////////////////////////
//
// SPMM Foward Pass (GAT, AGNN)
//
////////////////////////////////////////////
std::vector<torch::Tensor> spmmGAT_forward_cuda(
    torch::Tensor nodePointer,
    torch::Tensor edgeList,
    torch::Tensor edgeAttention,        //*edge attention [n_head, n_e]
    torch::Tensor blockPartition, 
    torch::Tensor edgeToColumn,
    torch::Tensor edgeToRow,
              int num_nodes,
              int num_edges,
              int embedding_dim,
    torch::Tensor input
) 
{
    auto output = torch::zeros_like(input);
    const int num_row_windows = blockPartition.size(0);
    const int WARPperBlock = WPB;
    const int num_attention = edgeAttention.size(0);
 
    hipStream_t* streams = new hipStream_t[num_attention];
    dim3 grid(num_row_windows, 1, 1);
    dim3 block(WARP_SIZE, WARPperBlock, 1);

    const int dimTileNum = (embedding_dim + BLK_H - 1) / BLK_H;
	const int dynamic_shared_size = dimTileNum * BLK_W * BLK_H * sizeof(float); // dynamic shared memory.

    // printf("spmmGAT_forward_cuda--1\n");
    for (int att_idx = 0; att_idx < num_attention; att_idx++){
        hipStreamCreate ( &streams[att_idx]) ;
        spmmGAT_forward_cuda_kernel<<<grid, block, dynamic_shared_size, streams[att_idx]>>>(
                                                                                        nodePointer.data<int>(), 
                                                                                        edgeList.data<int>(),
                                                                                        edgeAttention.data<float>(),
                                                                                        blockPartition.data<int>(), 
                                                                                        edgeToColumn.data<int>(), 
                                                                                        edgeToRow.data<int>(), 
                                                                                        num_nodes,
                                                                                        num_edges,
                                                                                        embedding_dim,
                                                                                        input.data<float>(), 
                                                                                        output.data<float>()
                                                                                    );
    }
    // printf("spmmGAT_forward_cuda--2\n");
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    return {output};
}

////////////////////////////////////////////
//
// SDDMM Foward Pass 
//
////////////////////////////////////////////
std::vector<torch::Tensor> sddmm_forward_cuda(
    torch::Tensor nodePointer,
    torch::Tensor edgeList,			    // edge list.
	torch::Tensor blockPartition,		// number of TC_blocks (16x8) in each row_window.
	torch::Tensor edgeToColumn, 		// eid -> col within each row_window.
	torch::Tensor edgeToRow, 			// eid -> col within each row_window.
              int num_nodes,
              int num_edges,
              int embedding_dim,	    // embedding dimension.
	torch::Tensor input				    // input feature matrix.
) 
{
    auto output = torch::zeros_like(edgeList).to(torch::kFloat);
    const int num_row_windows = blockPartition.size(0);

	dim3 grid(num_row_windows, 1, 1);
	dim3 block(WARP_SIZE, 1, 1);
    // printf("at sddmm_forward_cuda\n");

    sddmm_forward_cuda_kernel<<< grid, block>>>(
                                                nodePointer.data<int>(), 
                                                edgeList.data<int>(),
                                                blockPartition.data<int>(), 
                                                edgeToColumn.data<int>(), 
                                                edgeToRow.data<int>(), 
                                                num_nodes,
                                                num_edges,
                                                embedding_dim,
                                                input.data<float>(), 
                                                output.data<float>()
                                                );

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    return {output};
}



/*----------- CUDA Kernel ---------*/ 

////////////////////////////////////
/// SPMM forward (GCN, GraphSage)
///////////////////////////////////
__global__ void spmm_forward_cuda_kernel(
	const int * __restrict__ nodePointer,		// node pointer.
	const int *__restrict__ edgeList,			// edge list.
	const int *__restrict__ blockPartition, 	// number of TC_blocks (16x8) in each row_window.
	const int *__restrict__ edgeToColumn, 		// eid -> col within each row_window.
	const int *__restrict__ edgeToRow, 		    // eid -> col within each row_window.
	const int numNodes,
	const int numEdges,
	const int embedding_dim,				    // embedding dimension.
	const float *__restrict__ input,		    // input feature matrix.
	float *output							    // aggregated output feature matrix.
) {
    const unsigned bid = blockIdx.x;								// block_index == row_window_index
	const unsigned wid = threadIdx.y;								// warp_index handling multi-dimension > 16.
	const unsigned laneid = threadIdx.x;							// lanid of each warp.
	const unsigned tid = threadIdx.y * blockDim.x + laneid;			// threadid of each block.
	const unsigned warpSize = blockDim.x;							// number of threads per warp.
	const unsigned threadPerBlock = blockDim.x * blockDim.y;		// number of threads per block.

	const unsigned dimTileNum = embedding_dim / BLK_H;              // number of tiles along the dimension
	const unsigned nIdx_start = bid * BLK_H;					    // starting nodeIdx of current row_window.
	const unsigned nIdx_end = min((bid + 1) * BLK_H, numNodes);		// ending nodeIdx of current row_window.
	
	const unsigned eIdx_start = nodePointer[nIdx_start];			// starting edgeIdx of current row_window.
	const unsigned eIdx_end = nodePointer[nIdx_end];				// ending edgeIdx of the current row_window.
	const unsigned num_TC_blocks = blockPartition[bid]; 			// number of TC_blocks of the current row_window.
	const unsigned dense_bound = numNodes * embedding_dim;

	__shared__ float sparse_A[BLK_H * BLK_W];					// row-major sparse matrix shared memory store.
	__shared__ int sparse_AToX_index[BLK_W];					// TC_block col to dense_tile row.
	// __shared__ float dense_X[dimTileNum * BLK_W * BLK_H];	// column-major dense tile [dimTileNum, BLK_W, BLK_H]
	extern __shared__ float dense_X[];

	wmma::fragment<wmma::matrix_a, BLK_H, BLK_H, BLK_W, wmma::precision::tf32, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, BLK_H, BLK_H, BLK_W, wmma::precision::tf32, wmma::col_major> b_frag;
	wmma::fragment<wmma::accumulator, BLK_H, BLK_H, BLK_W, float> acc_frag;
	wmma::fill_fragment(acc_frag, 0.0f);

	// Processing TC_blocks along the column dimension of Sparse A.
	for (unsigned i = 0; i < num_TC_blocks; i++){

		// Init A_colToX_row with dummy values.
		if (tid < BLK_W){
			sparse_AToX_index[tid] = numNodes + 1;
		}

		__syncthreads();

		// Init sparse_A with zero values.
		#pragma unroll
		for (unsigned idx = tid; idx < BLK_W * BLK_H; idx += threadPerBlock){
			sparse_A[idx] = 0;
		}

		// Init dense_X with zero values.
		#pragma unroll
		for (unsigned idx = tid; idx < dimTileNum * BLK_W * BLK_H; idx += threadPerBlock){
			dense_X[idx] = 0;
		}

		// Initialize sparse_A by using BLK_H (16) threads from the warp-0.
		// currently fetch all neighbors of the current nodes.
		// then to see whether it can fit into current TC_block frame of column.		
		#pragma unroll
		for (unsigned eIdx = eIdx_start + tid; eIdx < eIdx_end; eIdx += threadPerBlock){
			unsigned col = edgeToColumn[eIdx];
			if (i * BLK_W <= col && col < (i + 1) * BLK_W){			// if the edge in the current TC_block frame of column.
				unsigned row_local = edgeToRow[eIdx] % BLK_H;
				unsigned col_local = col % BLK_W;
				sparse_A[row_local * BLK_W + col_local] = 1;		// set the edge of the sparse_A.
				sparse_AToX_index[col_local] = edgeList[eIdx];		// record the mapping from sparse_A colId to rowId of dense_X.
			}		
		}

		__syncthreads();

		// Initialize dense_X by column-major store,
		// Threads of a warp for fetching a dense_X.
		// each warp identify by wid.
		if (wid < dimTileNum)
			#pragma unroll
			for (unsigned idx = laneid; idx < BLK_W * BLK_H; idx += warpSize){
				unsigned dense_rowIdx = sparse_AToX_index[idx % BLK_W];						// TC_block_col to dense_tile_row.
				unsigned dense_dimIdx = idx / BLK_W;										// dimIndex of the dense tile.
				unsigned source_idx = dense_rowIdx * embedding_dim + wid * BLK_H + dense_dimIdx;
				unsigned target_idx = wid * BLK_W * BLK_H + idx;
				// boundary test.
				if (source_idx >= dense_bound)
					dense_X[target_idx] = 0;
				else
					dense_X[target_idx] = input[source_idx];
			}

		__syncthreads();

		if (wid < dimTileNum)
		{
			wmma::load_matrix_sync(a_frag, sparse_A, BLK_W);
			wmma::load_matrix_sync(b_frag, dense_X + wid * BLK_W * BLK_H, BLK_W);

			#pragma unroll
			for (unsigned t = 0; t < a_frag.num_elements; t++) {
				a_frag.x[t] =  wmma::__float_to_tf32(a_frag.x[t]);
			}

			#pragma unroll
			for (unsigned t = 0; t < b_frag.num_elements; t++) {
				b_frag.x[t] =  wmma::__float_to_tf32(b_frag.x[t]);
			}
			// Perform the matrix multiplication.
			wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
		}
	}

	if (wid < dimTileNum)
		// Store the matrix to output matrix.
		// * Note * embeeding dimension should be padded divisible by BLK_H for output correctness.
		wmma::store_matrix_sync(output + bid * BLK_H * embedding_dim + wid * BLK_H, acc_frag, embedding_dim, wmma::mem_row_major);
}

////////////////////////////////////
/// SPMM forward (GAT, AGNN)
///////////////////////////////////
__global__ void spmmGAT_forward_cuda_kernel(
	const int * __restrict__    nodePointer,		// node pointer.
	const int *__restrict__     edgeList,			// edge list.
    const float *__restrict__   edgeAttention,	    // edge attention.
	const int *__restrict__     blockPartition, 	// number of TC_blocks (16x8) in each row_window.
	const int *__restrict__     edgeToColumn, 		// eid -> col within each row_window.
	const int *__restrict__     edgeToRow, 		    // eid -> col within each row_window.
	const int numNodes,
	const int numEdges,
	const int embedding_dim,				    // embedding dimension.
	const float *__restrict__   input,		    // input feature matrix.
	float *output							    // aggregated output feature matrix.
) {
    const unsigned bid = blockIdx.x;								// block_index == row_window_index
	const unsigned wid = threadIdx.y;								// warp_index handling multi-dimension > 16.
	const unsigned laneid = threadIdx.x;							// lanid of each warp.
	const unsigned tid = threadIdx.y * blockDim.x + laneid;			// threadid of each block.
	const unsigned warpSize = blockDim.x;							// number of threads per warp.
	const unsigned threadPerBlock = blockDim.x * blockDim.y;		// number of threads per block.

	const unsigned dimTileNum = embedding_dim / BLK_H;              // number of tiles along the dimension
	const unsigned nIdx_start = bid * BLK_H;					    // starting nodeIdx of current row_window.
	const unsigned nIdx_end = min((bid + 1) * BLK_H, numNodes);		// ending nodeIdx of current row_window.
	
	const unsigned eIdx_start = nodePointer[nIdx_start];			// starting edgeIdx of current row_window.
	const unsigned eIdx_end = nodePointer[nIdx_end];				// ending edgeIdx of the current row_window.
	const unsigned num_TC_blocks = blockPartition[bid]; 			// number of TC_blocks of the current row_window.
	const unsigned dense_bound = numNodes * embedding_dim;

	__shared__ float sparse_A[BLK_H * BLK_W];					// row-major sparse matrix shared memory store.
	__shared__ int sparse_AToX_index[BLK_W];					// TC_block col to dense_tile row.
	// __shared__ float dense_X[dimTileNum * BLK_W * BLK_H];	// column-major dense tile [dimTileNum, BLK_W, BLK_H]
	extern __shared__ float dense_X[];

	wmma::fragment<wmma::matrix_a, BLK_H, BLK_H, BLK_W, wmma::precision::tf32, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, BLK_H, BLK_H, BLK_W, wmma::precision::tf32, wmma::col_major> b_frag;
	wmma::fragment<wmma::accumulator, BLK_H, BLK_H, BLK_W, float> acc_frag;
	wmma::fill_fragment(acc_frag, 0.0f);

	// Processing TC_blocks along the column dimension of Sparse A.
	for (unsigned i = 0; i < num_TC_blocks; i++){

		// Init A_colToX_row with dummy values.
		if (tid < BLK_W){
			sparse_AToX_index[tid] = numNodes + 1;
		}

		__syncthreads();

		// Init sparse_A with zero values.
		#pragma unroll
		for (unsigned idx = tid; idx < BLK_W * BLK_H; idx += threadPerBlock){
			sparse_A[idx] = 0;
		}

		// Init dense_X with zero values.
		#pragma unroll
		for (unsigned idx = tid; idx < dimTileNum * BLK_W * BLK_H; idx += threadPerBlock){
			dense_X[idx] = 0;
		}

		// Initialize sparse_A by using BLK_H (16) threads from the warp-0.
		// currently fetch all neighbors of the current nodes.
		// then to see whether it can fit into current TC_block frame of column.		
		#pragma unroll
		for (unsigned eIdx = eIdx_start + tid; eIdx < eIdx_end; eIdx += threadPerBlock){
			unsigned col = edgeToColumn[eIdx];
			if (i * BLK_W <= col && col < (i + 1) * BLK_W){			// if the edge in the current TC_block frame of column.
				unsigned row_local = edgeToRow[eIdx] % BLK_H;
				unsigned col_local = col % BLK_W;
				sparse_A[row_local * BLK_W + col_local] = edgeAttention[eIdx];		// sparse_A according to edge_features.
				sparse_AToX_index[col_local] = edgeList[eIdx];		                // record the mapping from sparse_A colId to rowId of dense_X.
			}		
		}

		__syncthreads();

		// Initialize dense_X by column-major store,
		// Threads of a warp for fetching a dense_X.
		// each warp identify by wid.
		if (wid < dimTileNum)
			#pragma unroll
			for (unsigned idx = laneid; idx < BLK_W * BLK_H; idx += warpSize){
				unsigned dense_rowIdx = sparse_AToX_index[idx % BLK_W];						// TC_block_col to dense_tile_row.
				unsigned dense_dimIdx = idx / BLK_W;										// dimIndex of the dense tile.
				unsigned source_idx = dense_rowIdx * embedding_dim + wid * BLK_H + dense_dimIdx;
				unsigned target_idx = wid * BLK_W * BLK_H + idx;
				// boundary test.
				if (source_idx >= dense_bound)
					dense_X[target_idx] = 0;
				else
					dense_X[target_idx] = input[source_idx];
			}

		__syncthreads();

		if (wid < dimTileNum)
		{
			wmma::load_matrix_sync(a_frag, sparse_A, BLK_W);
			wmma::load_matrix_sync(b_frag, dense_X + wid * BLK_W * BLK_H, BLK_W);

			#pragma unroll
			for (unsigned t = 0; t < a_frag.num_elements; t++) {
				a_frag.x[t] =  wmma::__float_to_tf32(a_frag.x[t]);
			}

			#pragma unroll
			for (unsigned t = 0; t < b_frag.num_elements; t++) {
				b_frag.x[t] =  wmma::__float_to_tf32(b_frag.x[t]);
			}
			// Perform the matrix multiplication.
			wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
		}
	}

	if (wid < dimTileNum)
		// Store the matrix to output matrix.
		// * Note * embeeding dimension should be padded divisible by BLK_H for output correctness.
		wmma::store_matrix_sync(output + bid * BLK_H * embedding_dim + wid * BLK_H, acc_frag, embedding_dim, wmma::mem_row_major);
}


//////////////////////
/// SDDMM forward
//////////////////////
__global__ void sddmm_forward_cuda_kernel(
    const int *__restrict__ nodePointer,
	const int *__restrict__ edgeList,			// edge list.
	const int *__restrict__ blockPartition,		// number of TC_blocks (16x8) in each row_window.
	const int *__restrict__ edgeToColumn, 		// eid -> col within each row_window.
	const int *__restrict__ edgeToRow, 			// eid -> col within each row_window.
	const int numNodes,
	const int numEdges,
	const int embedding_dim,				    // embedding dimension.
	float *__restrict__ in_mat,					// input feature matrix.
	float *edgeFeature							// aggregated output feature matrix.
)
{
    // printf("at sddmm_forward_cuda_kernel\n");
    unsigned bid = blockIdx.x;										// block_index == row_window_index
    unsigned wid = threadIdx.y;										// warp_index handling multi-dimension > 16.
    unsigned laneid = threadIdx.x;									// lanid of each warp.
    unsigned tid = threadIdx.y * blockDim.x + laneid;				// threadid of each block.

    unsigned threadPerBlock = blockDim.x * blockDim.y;
    unsigned DimIterations =  (embedding_dim + BLK_W - 1) / BLK_W; 	// dimension iteration for output.

    unsigned nid_start = bid * BLK_H;								// starting node_id of current row_window.
    unsigned nid_end = min((bid + 1) * BLK_H, numNodes);			// ending node_id of the current row_window.

    unsigned eIdx_start = nodePointer[nid_start];					            // starting eIdx of current row_window.
    unsigned eIdx_end = nodePointer[nid_end];						            // ending eIdx of the current row_window.
    unsigned num_TC_blocks = (blockPartition[bid] * BLK_W + BLK_H - 1)/BLK_H; 	// number of TC_blocks of the current row_window.

    __shared__ float sparse_A[BLK_H * BLK_H];					// 16 x 16 output sparse matrix.
    __shared__ float sparse_A_val[BLK_H * BLK_H];				// 16 x 16 output sparse matrix.
    #ifdef verify
    __shared__ float verify_A[BLK_H * BLK_H];					// 16 x 16 output sparse matrix.
    #endif 

    __shared__ unsigned sparse_AToX_index[BLK_H];				// TC_block col to dense_tile row.
    __shared__ float dense_X[BLK_H * BLK_W];
    __shared__ float dense_Y[BLK_W * BLK_H];

    wmma::fragment<wmma::matrix_a, BLK_H, BLK_H, BLK_W, wmma::precision::tf32, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, BLK_H, BLK_H, BLK_W, wmma::precision::tf32, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, BLK_H, BLK_H, BLK_W, float> acc_frag;
    wmma::fill_fragment(acc_frag, 0.0f);

    // Processing TC_blocks along the column dimension of Sparse A.
    // The block step here is 2, which is 16 = 8 + 8. 
    // In order to reuse the edgeToColumn in SpMM. 
    for (unsigned i = 0; i < num_TC_blocks; i++ ){

        if (wid == 0 && laneid < BLK_H){
            sparse_AToX_index[laneid] = numNodes + 1;
        }

        __syncthreads();

        #pragma unroll
        for (unsigned idx = tid; idx < BLK_H * BLK_H; idx += threadPerBlock){
            sparse_A[idx] = numEdges + 1;
            sparse_A_val[idx] = 0.0f;
        }

        #pragma unroll
        for (unsigned idx = tid; idx < BLK_W * BLK_H; idx += threadPerBlock){
            dense_X[idx] = 0;
            dense_Y[idx] = 0;
        }

        // Initialize sparse_A by using BLK_H (16) threads from the warp-0.
        // currently fetch all neighbors of the current nodes.
        // then to see whether it can fit into current TC_block frame of column.
        #pragma unroll
        // if (tid < WARP_SI)
        for (unsigned eIdx = tid + eIdx_start; eIdx < eIdx_end; eIdx += threadPerBlock){
            unsigned col = edgeToColumn[eIdx];						// condensed column id in sparse_A.
            if (i * BLK_H <= col && col < (i + 1) * BLK_H){			// if the edge in the current TC_block frame of column.
                unsigned row = edgeToRow[eIdx] % BLK_H;				// reverse indexing the row Id of the edge.
                sparse_A[row * BLK_H + col % BLK_H] = eIdx;			// set the edge of the sparse_A.
                sparse_AToX_index[col % BLK_H] = edgeList[eIdx];	// record the mapping from sparse_A colId to rowId of dense_X.
            }
        }		

        __syncthreads();

        for (unsigned warp_iter = 0; warp_iter < DimIterations; warp_iter++){
            // Initialize dense_X by row-major store,
            // Threads of a warp for fetching a dense_X.
            #pragma unroll
            for (unsigned i = tid; i < BLK_H * BLK_W; i += threadPerBlock){
                unsigned dense_rowIdx = i / BLK_W;					
                unsigned dense_dimIdx = i % BLK_W;					
                unsigned target_idx = i;
                unsigned source_idx = (nid_start + dense_rowIdx) * embedding_dim + warp_iter * BLK_W + dense_dimIdx;
                if (source_idx >= numNodes * embedding_dim)
                    dense_X[target_idx] = 0;
                else
                    dense_X[target_idx] = in_mat[source_idx];
            }

            // Initialize dense_Y by column-major store,
            // Threads of a warp for fetching a dense_Y.
            #pragma unroll
            for (unsigned i = tid; i < BLK_W * BLK_H; i += threadPerBlock){
                unsigned dense_rowIdx = sparse_AToX_index[i / BLK_W];					// TC_block_col to dense_tile_row.
                unsigned dense_dimIdx = i % BLK_W;										// dimIndex of the dense tile.
                unsigned target_idx = i;
                unsigned source_idx = dense_rowIdx * embedding_dim + warp_iter * BLK_W + dense_dimIdx;
                if (source_idx >= numNodes * embedding_dim)
                    dense_Y[target_idx] = 0;
                else
                    dense_Y[target_idx] = in_mat[source_idx];
            }

            __syncthreads();

            wmma::load_matrix_sync(a_frag, dense_X, BLK_W);
            wmma::load_matrix_sync(b_frag, dense_Y, BLK_W);

            #pragma unroll
            for (unsigned t = 0; t < a_frag.num_elements; t++) {
                a_frag.x[t] =  wmma::__float_to_tf32(a_frag.x[t]);
            }

            #pragma unroll
            for (unsigned t = 0; t < b_frag.num_elements; t++) {
                b_frag.x[t] =  wmma::__float_to_tf32(b_frag.x[t]);
            }

            // Perform the matrix multiplication on Tensor Core
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);		
        } // <--- ending of warp iteration.

        wmma::store_matrix_sync(sparse_A_val, acc_frag, BLK_H, wmma::mem_row_major);
        wmma::fill_fragment(acc_frag, 0.0f);

        // Output the results to sparse matrix edge featureList.
        for (unsigned t = 0; t < BLK_H * BLK_H; t++) {
            unsigned rowId = t / BLK_H;
            unsigned colId = t % BLK_H;
            if (sparse_A[rowId * BLK_H + colId] < numEdges){
                unsigned eIdx = sparse_A[rowId * BLK_H + colId];
                edgeFeature[eIdx] = sparse_A_val[t];
            }
        } //<-- ending of storing output to global memory.
    }
}